#include "hip/hip_runtime.h"
#include"potential_coeff.h"

__global__ void get_potential_matrix_elements(float *impurity_x,float *impurity_y,float *impurity_intensity, hipFloatComplex* Vmn, hipFloatComplex* coeff_mn, int dim_m, int dim_n, int off_head, int impurity_num){
    int m=blockIdx.x*blockDim.x+threadIdx.x;
    int n=blockIdx.y*blockDim.y+threadIdx.y;
    if(m<dim_m && n<dim_n){
    float kx=(m-off_head)*PI2;
    float ky=(n-off_head)*PI2;
    float phase;
    hipFloatComplex sum;
    sum.x=sum.y=0;
    for(int i=0;i<impurity_num;i++){
       phase=kx*impurity_x[i]+ky*impurity_y[i];
       sum.x+=cos(phase)*impurity_intensity[i];
       sum.y+=sin(phase)*impurity_intensity[i];
    }
    Vmn[m*dim_n+n]=hipCmulf(sum,coeff_mn[m*dim_n+n]);
    }
}

void prepare_potential_coeff(float* impurity_x, float * impurity_y, float *impurity_intensity, hipFloatComplex * Vmn, hipFloatComplex* coeff_mn, int dim_m, int dim_n, int off_head,int impurity_num) {
    dim3 grids(dim_m/4+1,dim_n/4+1);
    dim3 blocks(4,4);
    get_potential_matrix_elements<<<grids,blocks,0,NULL>>>(impurity_x, impurity_y,impurity_intensity,Vmn,coeff_mn,dim_m,dim_n,off_head, impurity_num);
    hipStreamSynchronize(NULL);
}
