#include "hip/hip_runtime.h"
#include"hamiltonian_gpu.h"
__device__ __host__ hipFloatComplex  operator*(hipFloatComplex a,hipFloatComplex b) {
    return hipCmulf(a,b);
}
__device__ __host__ hipFloatComplex  operator+(hipFloatComplex a,hipFloatComplex b) {
    return hipCaddf(a,b);
}

__global__ void get_matrix_elements(const hipFloatComplex* coeff_jm, const hipFloatComplex* coeff_m_theta_1,const hipFloatComplex* coeff_m_theta_2, const hipFloatComplex* Vmn, hipFloatComplex* wfs, int dim_n, int off_head) {
    int nphi=gridDim.x*blockDim.x;
    int j=blockIdx.x*blockDim.x+threadIdx.x;
    int k=blockIdx.y*blockDim.y+threadIdx.y;
    hipFloatComplex sum;
    int wfs_idx=j*nphi+k;
    int idx_m,n,m;
    wfs[wfs_idx].x=wfs[wfs_idx].y=0;
    if(k>=j) {
        for( m = 0; m < dim_n; m++) {
            sum.x=sum.y=0;
            idx_m=m*dim_n;
            if(k-j<=off_head ) {
                n=off_head+k-j;
                sum=sum+Vmn[idx_m + n]*hipConjf(coeff_m_theta_1[n]);
            }
            if((nphi+j-k)<=off_head) {
                n=off_head+k-j-nphi;
                sum=sum+Vmn[idx_m + n]*hipConjf(coeff_m_theta_1[n]);
            }
            // old version of Kronecker delta
            /*
                for(int n = 0; n < dim_n; n++)
                    if(abs(j-k+n-off_head)%nphi==0) {
                        sum=sum+Vmn[idx_m + n]*coeff_m_theta_1[n];
                    }
            */
            wfs[wfs_idx]= wfs[wfs_idx]+ sum* coeff_jm[j*dim_n+m]*coeff_m_theta_2[m];
        }
        wfs[k*nphi+j]=hipConjf(wfs[wfs_idx]);
    }

}

void set_hamil_matrix(const hipFloatComplex* dev_coeff_m_theta_1,const hipFloatComplex* dev_coeff_m_theta_2, const hipFloatComplex* dev_coeff_jm, const hipFloatComplex *dev_Vmn, hipFloatComplex* dev_wfs, int n_phi,int dim_n,int off_head){
    // n_phi should be multiplier of 8 
    dim3 grids(n_phi/8,n_phi/8);
    dim3 blocks(8,8);
    // calculating the hamiltonian matrix elements
    get_matrix_elements<<<grids,blocks,0,NULL>>>(dev_coeff_jm,dev_coeff_m_theta_1,dev_coeff_m_theta_2,dev_Vmn,dev_wfs,dim_n,off_head);
    hipStreamSynchronize(NULL);
}

