#include "hip/hip_runtime.h"
#include<iostream>
#include<fstream>
#include<iomanip>
#include<complex>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include"init_rlm.h"
#include"matrix_coefficients.h"
#include"disorder_potential.h"
#include"potential_coeff.h"
#include"hamiltonian_rlm_gpu.h"
#include"wfs_file.h"
#include"chern.h"
// macros for using different
// diagonalization routines
#if defined magma
#include"magma_diag.h"
#elif defined mkl
#include "mkl_diag.h"
#elif defined cusolver
#include"cusolver_diag.h"
#endif

using namespace std;
int main(int argc, char *argv[]) {
    /************************************  PARAMETERS INITIALIZATION  *******************************************/
    long n_phi,n_mesh,num_threads,n_sample,impurity_num,off_head, dim_m,dim_n;
    unsigned long seed,dim_wfs;
    float quanta_concentration,impurity_concentration,L1,L2;
    init(argc,argv,n_phi,quanta_concentration,impurity_concentration,n_mesh,n_sample,seed,num_threads);
    L1=sqrt(n_phi*quanta_concentration);
    L2=sqrt(n_phi*quanta_concentration);
    impurity_num = (impurity_concentration * n_phi / quanta_concentration);
    off_head =sqrt(impurity_num)/2+1;
    dim_m =dim_n= off_head * 2 + 1;
    dim_wfs=n_phi*n_phi;
    

    /*************************************  HOST MEMORY ALLOCATION    ******************************************/
    // Coefficients for calculating hamiltonian--> disorder independent coefficients
    complex<float> *coeff_mn, *coeff_m_theta, *coeff_jm;
    // disorder dependent coefficients
    // impurity positions
    float *impurity_x, *impurity_y;
    // impurity intensities
    float *impurity_intensity;
    // wave functions for all theta_1, theta_2
    complex<float> *wfs_full;
    // theta_1, theta_2 averaged eigenvalues
    float *energy_theta;
    // page-locked host memory, for data transfer
    // if use wfs write and read, reduces wfs in memory to 1*dim_wfs, otherwise 2*(n_mesh+1)*dim_wfs
    #ifdef wfsIO
    hipHostAlloc((void**)&wfs_full,dim_wfs*sizeof(*wfs_full),hipHostMallocMapped);
    #else
    unsigned long wfs_size=(n_mesh+1)*2*dim_wfs*sizeof(complex<float>);
    hipHostAlloc((void**)&wfs_full,wfs_size,hipHostMallocMapped);
    #endif
    hipHostAlloc((void**)&coeff_m_theta,(n_mesh+1)*dim_m*sizeof(*coeff_m_theta),hipHostMallocDefault);
    hipHostAlloc((void**)&coeff_jm,n_phi*dim_m*sizeof(*coeff_jm),hipHostMallocDefault);
    hipHostAlloc((void**)&energy_theta,(n_mesh+1)*n_phi*sizeof(*energy_theta),hipHostMallocDefault);

    coeff_mn=new complex<float>[dim_m*dim_n];
    impurity_x = new float[impurity_num];
    impurity_y = new float[impurity_num];
    impurity_intensity = new float[impurity_num];

    float *energy_levels = new float[n_phi];
    float *chern_numbers = new float[n_phi];
    float *chern_numbers_theta = new float[n_mesh *n_phi];

    /*************************************  DEVICE MEMORY ALLOCATION    ******************************************/
    complex<float> *dev_wfs;
    complex<float> *dev_coeff_m_theta_1;
    complex<float> *dev_coeff_m_theta_2;
    complex<float> *dev_coeff_jm;
    complex<float> *dev_coeff_mn;
    complex<float> *dev_Vmn;
    float *dev_energy;

    float * dev_impurity_x;
    float * dev_impurity_y;
    float * dev_impurity_intensity;


    // allocate device memory
    hipMalloc ((void**)&dev_wfs,n_phi*n_phi*sizeof(*dev_wfs));
    hipMalloc ((void**)&dev_coeff_m_theta_1,dim_m*sizeof(*dev_coeff_m_theta_1));
    hipMalloc ((void**)&dev_coeff_m_theta_2,dim_m*sizeof(*dev_coeff_m_theta_2));
    hipMalloc ((void**)&dev_coeff_jm,n_phi*dim_m*sizeof(*dev_coeff_jm));
    hipMalloc ((void**)&dev_coeff_mn,dim_n*dim_m*sizeof(*dev_coeff_mn));
    hipMalloc ((void**)&dev_Vmn,dim_m*dim_n*sizeof(*dev_Vmn));
    hipMalloc ((void**)&dev_energy,n_phi*sizeof(*dev_energy));
    hipMalloc ((void**)&dev_impurity_x,impurity_num*sizeof(*dev_impurity_x));
    hipMalloc ((void**)&dev_impurity_y,impurity_num*sizeof(*dev_impurity_y));
    hipMalloc ((void**)&dev_impurity_intensity,impurity_num*sizeof(*dev_impurity_intensity));


    /************************************* PTHREAD PARAMETERS  **************************************************/
    int **thds_ctheta;
    int *ctheta_len;
    // theta threads allocate
    thds_ctheta = new int*[num_threads];
    for(int i = 0; i < num_threads; i++) {
        thds_ctheta[i] = new int[n_mesh / num_threads + 1];
    }
    ctheta_len = new int[num_threads];

    // divide works into num_threads of pieces
    memset(ctheta_len, 0, num_threads*sizeof(int));
    for(int i = 0; i < n_mesh; i++)
        thds_ctheta[i% num_threads][ctheta_len[i%num_threads]++]=i;
    pthread_t *peer_thds = new pthread_t[num_threads];

    hipEvent_t start,stop;
    float gpu_time,pot_time,coeff_time,chern_time,total_time;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    /***********************************  COEFFICIENTS INITIALIZATION  *******************************************/
    hipEventRecord(start,0);
    // initialize and store the disorder independent coefficients
    prepare_coeff(coeff_mn, coeff_m_theta, coeff_jm,n_phi,off_head,dim_m,dim_n,n_mesh, L1, L2);
    // copy coefficients from host to device
    hipMemcpy(dev_coeff_jm,coeff_jm,sizeof(*coeff_jm)*n_phi*dim_m,hipMemcpyHostToDevice);
    hipMemcpy(dev_coeff_mn,coeff_mn,sizeof(*coeff_mn)*dim_n*dim_m,hipMemcpyHostToDevice);
    // initialize data_file output
    ofstream fchern;
    fchern.open("energy_Chern_No.dat");

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&coeff_time,start,stop);
    coeff_time/=1000.0;
    cerr<<std::left<<setw(40)<<"coefficients initialization time: "<<coeff_time<<" s \n";

    /*********************************** MAIN PROGRAM ************************************************************/
    for(int n = 0; n < n_sample; n++) {
        memset(energy_levels, 0, n_phi*sizeof(float));
        memset(chern_numbers, 0, n_phi*sizeof(float));

        hipEventRecord(start,0);
        // initialize the disorder potential
        generate_disorder_potential(impurity_x,impurity_y,impurity_intensity,impurity_num,L1,L2,seed,n);
        hipMemcpy(dev_impurity_x,impurity_x,sizeof(*impurity_x)*impurity_num,hipMemcpyHostToDevice);
        hipMemcpy(dev_impurity_y,impurity_y,sizeof(*impurity_y)*impurity_num,hipMemcpyHostToDevice);
        hipMemcpy(dev_impurity_intensity,impurity_intensity,
		sizeof(*impurity_intensity)*impurity_num,hipMemcpyHostToDevice);
        // calculate the coefficient part from potential and transfer to the device
	prepare_potential_coeff(dev_impurity_x, dev_impurity_y, dev_impurity_intensity,
			(hipFloatComplex*)dev_Vmn,(hipFloatComplex*)dev_coeff_mn,dim_m,dim_n,off_head,impurity_num);

        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&pot_time,start,stop);
	pot_time/=1000.0;
	if(n==0)
          cerr<<setw(40)<<"potential initialization time: "<<pot_time<<" s\n";

        // solve the (theta_1=0)-line of wave functions
        int theta_1=0;
        for(int theta_2=0; theta_2<=n_mesh; theta_2++) {
	    int wfs_index;
            #if defined wfsIO
	    wfs_index = 0;
            #else 
            wfs_index = ((theta_1%2) * (n_mesh + 1) + theta_2) * dim_wfs;
	    #endif
            int en_index = theta_2 * n_phi;
            hipMemcpy(dev_coeff_m_theta_1,coeff_m_theta+theta_1*dim_n,dim_n*sizeof(*coeff_m_theta),hipMemcpyHostToDevice);
            hipMemcpy(dev_coeff_m_theta_2,coeff_m_theta+theta_2*dim_n,dim_n*sizeof(*coeff_m_theta),hipMemcpyHostToDevice);
            set_hamil_matrix((hipFloatComplex*)dev_coeff_m_theta_1,(hipFloatComplex*)dev_coeff_m_theta_2,
                             (hipFloatComplex*) dev_coeff_jm,(hipFloatComplex*)dev_Vmn, (hipFloatComplex*)dev_wfs,
                             n_phi,dim_n,off_head);
            #if defined magma
            magma_diag_hamil((magmaFloatComplex*)dev_wfs,energy_theta+en_index,n_phi);
            hipMemcpy(wfs_full+wfs_index, dev_wfs, sizeof(*(dev_wfs))*dim_wfs, hipMemcpyDeviceToHost);
            #elif defined mkl
            hipMemcpy(wfs_full+wfs_index, dev_wfs, sizeof(*(dev_wfs))*dim_wfs, hipMemcpyDeviceToHost);
            mkl_heevd(wfs_full+wfs_index,energy_theta+en_index,n_phi);
            #elif defined cusolver
            cusolver_diag_hamil((hipFloatComplex*)dev_wfs,dev_energy,n_phi);
            hipMemcpy(wfs_full+wfs_index, dev_wfs, sizeof(*(dev_wfs))*dim_wfs, hipMemcpyDeviceToHost);
            hipMemcpy(energy_theta+en_index, dev_energy, sizeof(*(dev_energy))*n_phi, hipMemcpyDeviceToHost);
            #endif
            #if defined wfsIO
	    write_wfs(theta_1%2,theta_2,wfs_full,dim_wfs);
            #endif
        }

        // average the (theta_1=0)-line of energy over theta_2
        // energy will be divided by (n_mesh+1)*(n_mesh+1) finally
        for(int i=0; i<n_phi; i++) {
            for(int j=0; j<=n_mesh; j++)
                energy_levels[i] += energy_theta[j* n_phi +i];
        }
        // solve theta_1-line of wfs from 1 to n_mesh, chern number calculations are also performed
        for(int theta_1=1; theta_1<=n_mesh; theta_1++) {
            if(theta_1==1)
              hipEventRecord(start,0);
            // solve for another line
            for(int theta_2=0; theta_2<=n_mesh; theta_2++) {
		int wfs_index; 
                #if defined wfsIO
                wfs_index = 0;
                #else 
                wfs_index = ((theta_1%2) * (n_mesh + 1) + theta_2) * dim_wfs;
                #endif
                int en_index = theta_2 * n_phi;
                hipMemcpy(dev_coeff_m_theta_1,coeff_m_theta+theta_1*dim_n,dim_n*sizeof(*coeff_m_theta),hipMemcpyHostToDevice);
                hipMemcpy(dev_coeff_m_theta_2,coeff_m_theta+theta_2*dim_n,dim_n*sizeof(*coeff_m_theta),hipMemcpyHostToDevice);
                set_hamil_matrix((hipFloatComplex*)dev_coeff_m_theta_1,(hipFloatComplex*)dev_coeff_m_theta_2,
                                 (hipFloatComplex*) dev_coeff_jm,(hipFloatComplex*)dev_Vmn, (hipFloatComplex*)dev_wfs,
                                 n_phi,dim_n,off_head);
                #if defined magma
                magma_diag_hamil((magmaFloatComplex*)dev_wfs,energy_theta+en_index,n_phi);
                hipMemcpy(wfs_full+wfs_index, dev_wfs, sizeof(*(dev_wfs))*dim_wfs, hipMemcpyDeviceToHost);
                #elif defined mkl
                hipMemcpy(wfs_full+wfs_index, dev_wfs, sizeof(*(dev_wfs))*dim_wfs, hipMemcpyDeviceToHost);
                mkl_heevd(wfs_full+wfs_index,energy_theta+en_index,n_phi);
                #elif defined cusolver
                cusolver_diag_hamil((hipFloatComplex*)dev_wfs,dev_energy,n_phi);
                hipMemcpy(wfs_full+wfs_index, dev_wfs, sizeof(*(dev_wfs))*dim_wfs, hipMemcpyDeviceToHost);
                hipMemcpy(energy_theta+en_index, dev_energy, sizeof(*(dev_energy))*n_phi, hipMemcpyDeviceToHost);
                #endif
                #if defined wfsIO
	        write_wfs(theta_1%2,theta_2,wfs_full,dim_wfs);
                #endif
            }
            // average the energy
            for(int i=0; i<n_phi; i++) {
                for(int j=0; j<n_mesh+1; j++)
                    energy_levels[i] += energy_theta[j* n_phi +i];
            }
	    if(theta_1==1 && n==0){
              hipEventRecord(stop,0);
              hipEventSynchronize(stop);
              hipEventElapsedTime(&gpu_time,start,stop);
	      gpu_time/=1000.0;
              cerr<<setw(40)<<"diagonalization time/k-point: "<<gpu_time/(n_mesh+1)<<" s\n";
              hipEventRecord(start,0);
	    }
            #if defined wfsIO
            cal_Chern_wfs_IO(chern_numbers_theta,n_phi,n_phi, n_mesh, theta_1);
            #else
            // calculate Chern numbers for two lines
	    /*
            for(int id = 0; id < num_threads; id++) {
                peer_Chern_paramsT *params;
                params = (peer_Chern_paramsT *) malloc(sizeof(peer_Chern_paramsT));
                params-> n_phi = n_phi;
                params-> dim_vec = n_phi;
                params-> n_mesh = n_mesh;
                params-> theta_1 = theta_1;
                params-> theta_2 = thds_ctheta[id];
                params-> theta_len = ctheta_len[id];
                params-> wave_function = wfs_full;
                params-> chern_numbers_theta = chern_numbers_theta;
                pthread_create(&(peer_thds[id]), NULL, peer_cal_Chern,  (void*)params);
            }
            // join all the threads
            for(int id = 0; id < num_threads; id++)
                pthread_join(peer_thds[id], NULL);
		*/
            cal_Chern(wfs_full,chern_numbers_theta,n_phi,n_phi, n_mesh, theta_1);
            #endif
	    if(theta_1==1 && n==0){
              hipEventRecord(stop,0);
              hipEventSynchronize(stop);
              hipEventElapsedTime(&chern_time,start,stop);
	      chern_time/=1000.0;
	      total_time=coeff_time+(pot_time+(chern_time*n_mesh+gpu_time*(n_mesh+1)))*n_sample;
              cerr<<setw(40)<<"Chern No. calculation time/k-point: "<<chern_time/(n_mesh+1)<<" s\n";
	      cerr<<setw(40)<<"estimated total time:"<<std::right<<setw(3)<<setfill('0')<<(int(total_time))/3600<<setw(1)<<":";
	      cerr<<setw(2)<<setfill('0')<<((int(total_time))%3600)/60<<setw(1)<<":";
	      cerr<<setw(2)<<setfill('0')<<(int(total_time))%60<<endl<<setfill(' ');
	      cerr<<std::left<<setw(40)<<"estimated # of samples/hour:"<<n_sample/(total_time/3600.0)<<endl;
	      cerr<<setw(40)<<"# module "<<" %(time)"<<endl;
	      cerr<<setw(40)<<"coefficients"<<coeff_time/total_time*100.0<<"\n";
	      cerr<<setw(40)<<"potential"<<pot_time*n_sample/total_time*100.0<<"\n";
	      cerr<<setw(40)<<"diagonalization"<<gpu_time*(n_mesh+1)*n_sample/total_time*100.0<<endl;
	      cerr<<setw(40)<<"Chern No. calculation"<<chern_time*n_mesh*n_sample/total_time*100.0<<endl;
	    }
            //serial version
            //cal_Chern(wfs_full,chern_numbers_theta,n_phi,n_phi, n_mesh, theta_1);
            // collect the chern number contributions from the theta_1 line
            for(int k=0; k<n_phi; k++) {
                for(int i=0; i<n_mesh; i++)
                    chern_numbers[k]+=chern_numbers_theta[k*n_mesh+i];
            }
        }
        //average the energy levels by dividing (n_mesh+1)*(n_mesh+1) at the final stage
        for(int k=0; k<n_phi; k++) {
            energy_levels[k]/=(n_mesh+1)*(n_mesh+1);
            chern_numbers[k]+=1.0/n_phi;
        }
        // print averaged energy levels
        for(int i = 0; i < n_phi; i++)
            fchern << setw(8) << n << " " << setw(15) << setprecision(8) << energy_levels[i] << " " << chern_numbers[i] << endl;
    }

    /***********************************  CLEAR DEVICE MEMORY  ***************************************/
    hipFree(dev_wfs);
    hipFree(dev_coeff_jm);
    hipFree(dev_coeff_m_theta_1);
    hipFree(dev_coeff_m_theta_2);
    hipFree(dev_coeff_mn);
    hipFree(dev_Vmn);
    hipFree(dev_energy);
    hipFree(dev_impurity_x);
    hipFree(dev_impurity_y);
    hipFree(dev_impurity_intensity);

    /***********************************  CLEAR HOST MEMORY  ****************************************/
    delete[] chern_numbers;
    delete[] energy_levels;
    delete[] impurity_intensity;
    delete[] impurity_y;
    delete[] impurity_x;
    delete[] coeff_mn;

    hipHostFree(wfs_full);
    hipHostFree(coeff_jm);
    hipHostFree(coeff_m_theta);
    hipHostFree(energy_theta);

    fchern.close();

    return 0;
}
