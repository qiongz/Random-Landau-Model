#include"cusolver_diag.h"

void cusolver_diag_hamil(hipFloatComplex* dev_wfs, float*dev_energy,int n_phi){
    hipsolverHandle_t cusolverH ;
    hipsolverDnCreate (& cusolverH );
    hipsolverSetStream(cusolverH,NULL);
    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR;
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    hipFloatComplex*dev_work;
    int *info;
    int lwork;
    // querying workspace
    hipsolverDnCheevd_bufferSize (cusolverH,jobz, uplo, n_phi, dev_wfs, n_phi, dev_energy, & lwork );
    hipMalloc (( void **)& dev_work, sizeof (*dev_work)* lwork );
    hipMalloc((void**)&info, sizeof(*info));
    // diagonalization the hamiltonian
    hipsolverDnCheevd(cusolverH,jobz,uplo,n_phi,dev_wfs,n_phi,dev_energy,dev_work,lwork,info);
    hipStreamSynchronize(NULL);
    hipsolverDnDestroy(cusolverH);
    hipFree(dev_work);
    hipFree(info);
}

